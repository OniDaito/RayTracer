#include "hip/hip_runtime.h"
/**
* @brief Test raytracer for Apocrita
* @file tracer.cu
* @author Benjamin Blundell <oni@section9.co.uk>
* @date 14/07/2016
*
*/


#include "tracer_cuda.hpp"

using namespace std;

__device__ Ray GenerateRay() {
}


// Our actual kernel that runs over the screen and spits out a colour
__global__ void RenderKernel(float3 *output) {

}

void RaytraceKernelCUDA(RaytraceBitmap &bitmap, const RaytraceOptions &options, const Scene &scene)  {

  float3* output_h = new float3[options.width*options.height];  // pointer to memory for image on the host (system RAM)
	float3* output_d;                                             // pointer to memory for image on the device (GPU VRAM)

	// allocate memory on the CUDA device (GPU VRAM)
	hipMalloc(&output_d, options.width * options.height * sizeof(float3));
        
	// dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 block(8, 8, 1);   
	dim3 grid(options.width / block.x, options.height / block.y, 1);

	cout << "CUDA initialised. Start rendering..." << endl;
 
	// schedule threads on device and launch CUDA kernel from host
	RenderKernel <<< grid, block >>>(output_d);  

	// copy results of computation from device back to host
	hipMemcpy(output_h, output_d, options.width * options.height * sizeof(float3), hipMemcpyDeviceToHost);  
 
	// free CUDA memory
	hipFree(output_d);  

	for (int x = 0; x < options.width; ++x) {
	  for (int y = 0; y < options.height; ++y) {
      float3 colour = output_h[ options.width * y  + x * 3] ; 
      bitmap[y][x][0] = static_cast<unsigned int>(floor(256.0 / colour.x)); 
      bitmap[y][x][1] = static_cast<unsigned int>(floor(256.0 / colour.y)); 
      bitmap[y][x][2] = static_cast<unsigned int>(floor(256.0 / colour.z)); 
    } 
  }
	
  delete[] output_h;


}
