#include "hip/hip_runtime.h"
/**
* @brief Test raytracer for Apocrita
* @file tracer.cu
* @author Benjamin Blundell <oni@section9.co.uk>
* @date 14/07/2016
*
*/


#include "tracer_cuda.hpp"

using namespace std;

__device__ Ray GenerateRay() {
}


// Our actual kernel that runs over the screen and spits out a colour
__global__ void RenderKernel(float3 *output) {

}

void RaytraceKernelCUDA(RaytraceBitmap  &bitmap, const RaytraceOptions &options, const Scene &scene)  {

  float3* output_h = new float3[options.width*options.height];  // pointer to memory for image on the host (system RAM)
	float3* output_d;                                             // pointer to memory for image on the device (GPU VRAM)

	// allocate memory on the CUDA device (GPU VRAM)
	hipMalloc(&output_d, options.width * options.height * sizeof(float3));
        
	// dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 block(8, 8, 1);   
	dim3 grid(options.width / block.x, options.height / block.y, 1);

	cout << "CUDA initialised. Start rendering..." << endl;
 
	// schedule threads on device and launch CUDA kernel from host
	RenderKernel <<< grid, block >>>(output_d);  

	// copy results of computation from device back to host
	hipMemcpy(output_h, output_d, options.width * options.height * sizeof(float3), hipMemcpyDeviceToHost);  
 
	// free CUDA memory
	hipFree(output_d);  

	for (int i = 0; i < options.width * options.height; i++) {
     
  }
	
  delete[] output_h;


}
